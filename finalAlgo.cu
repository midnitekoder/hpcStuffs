
#include <hip/hip_runtime.h>
#include<stdio.h>

#define numKernels 512
#define hashTableWidth 12288
#define searchKernels 1024
#define numHashPerThread 24
#define numThreadsPerBlock 1


__device__ void swapTriplets(int *d_x,int *d_y,int *d_z,int *d_indexOrder, int index1,int index2)
{
	int temp1,temp2,temp3;
	int temp4;
	temp1=d_x[index1];
	temp2=d_y[index1];
	temp3=d_z[index1];
	temp4=d_indexOrder[index1];
	d_x[index1]=d_x[index2];
	d_y[index1]=d_y[index2];
	d_z[index1]=d_z[index2];
	d_indexOrder[index1]=d_indexOrder[index2];
	d_x[index2]=temp1;
	d_y[index2]=temp2;
	d_z[index2]=temp3;
	d_indexOrder[index2]=temp4;
}


__device__ int compare(int x1,int y1, int z1, int x2, int y2, int z2)
{
	if((x1==x2) && (y1==y2) && (z1==z2))
		return 0;
	if((x1>x2) ||((x1==x2) && (y1>y2)) || ((x1==x2) && (y1==y2) && (z1>z2)))
		return 1;
	return -1;
}

__global__ void createPartition(int count,int *d_x,int *d_y,int *d_z,int *d_indexOrder,int *d_partition_begin,int *d_partition_last)
{
	int partitionIndex=blockIdx.x*blockDim.x+threadIdx.x;
	int begin,last,i,j;
	int x,y,z;
	int indexOrder;
	begin=d_partition_begin[partitionIndex];
	last=d_partition_last[partitionIndex];
	if((begin>=0) && (last>=0))
	{
		if(begin<last)
		{
			swapTriplets(d_x,d_y,d_z, d_indexOrder
,last,((begin+last)/2));
			x=d_x[last];
			y=d_y[last];
			z=d_z[last];
			indexOrder=d_indexOrder[last];
			i = (begin - 1);
	
			for ( j = begin; j <= last- 1; j++)
			{
				if (compare(d_x[j],d_y[j],d_z[j],x,y,z)<=0)
				{
    					i++;
    					swapTriplets (d_x,d_y,d_z,d_indexOrder,i, j);
				}
			}
			swapTriplets (d_x,d_y,d_z,d_indexOrder,(i+1),(last));

		}
	}
	__syncthreads();
	if((begin>=0) && (last>=0))
	{
		if(begin<last)
		{
			d_partition_begin[2*partitionIndex]=begin;
			d_partition_last[2*partitionIndex]=i;
			d_partition_begin[2*partitionIndex+1]=i+1;
			d_partition_last[2*partitionIndex+1]=last;
		}
		else
		{
			d_partition_begin[2*partitionIndex]=begin;
			d_partition_last[2*partitionIndex]=last;
			d_partition_begin[2*partitionIndex+1]=-1;
			d_partition_last[2*partitionIndex+1]=-1;
		}
	}
	else
	{
		d_partition_begin[2*partitionIndex]=-1;
		d_partition_last[2*partitionIndex]=-1;
		d_partition_begin[2*partitionIndex+1]=-1;
		d_partition_last[2*partitionIndex+1]=-1;
	}
}



__global__ void countPartitions(int *d_partition_begin,int *d_partition_last,int *d_numPartitions)
{
	int i,j;
	int temp_begin,temp_last;
	j=-1;
	for(i=0;i<numKernels;i++)
	{
		if((d_partition_begin[i]!=-1)&&(d_partition_last[i]!=-1))
		{
			j++;
			temp_begin=d_partition_begin[i];
			temp_last=d_partition_last[i];
			d_partition_begin[i]=d_partition_begin[j];
			d_partition_last[i]=d_partition_last[j];
			d_partition_begin[j]=temp_begin;
			d_partition_last[j]=temp_last;

		}
	}
	d_numPartitions[0]=j+1;

}

__global__ void initializePartitionTable(int *d_partition_begin, int *d_partition_last)
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<numKernels)
	{
		d_partition_begin[index]=-1;
		d_partition_last[index]=-1;
	}
}


__device__ int binarySearch(int *d_x, int *d_y, int *d_z,int begin, int offset,int hashvalue)
{

	int mid,i;
	if(begin<=offset)
	{
		mid=(begin+offset)/2;
		if(((d_x[mid]+d_y[mid]+d_z[mid])%numHashPerThread)==hashvalue)
		{
			for(i=mid+1;i<=offset;i++)
			{
				if(((d_x[i]+d_y[i]+d_z[i])%numHashPerThread)>hashvalue)
					break;
			}
			return i;
		}
		else if((((d_x[mid]+d_y[mid]+d_z[mid])%numHashPerThread)<hashvalue) && (((mid<offset)&&(((d_x[mid+1]+d_y[mid+1]+d_z[mid+1])%numHashPerThread)>hashvalue))||(mid==offset)))
		{
			return mid;
		}
		else if (((d_x[mid]+d_y[mid]+d_z[mid])%numHashPerThread)>hashvalue)
			return binarySearch(d_x,d_y,d_z,begin,mid-1,hashvalue);
		else
			return binarySearch(d_x,d_y,d_z,mid+1, offset,hashvalue);
	}
	return -1;



}

__device__ void shiftPos(int *d_x, int *d_y, int *d_z,int *d_indexOrder, int offset)
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int x,y,z;
	int indexOrder;
	x=d_x[offset-index];
	y=d_y[offset-index];
	z=d_z[offset-index];
	indexOrder=d_indexOrder[offset-index];
	__syncthreads();
	d_x[offset-index+1]=x;
	d_y[offset-index+1]=y;
	d_z[offset-index+1]=z;
	d_indexOrder[offset-index+1]=indexOrder;
}


__global__ void setIndexOrder(int *d_indexOrder, int count)
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int i;
	for(i=index*((count+numKernels-1)/numKernels);i<(index+1)*((count+numKernels-1)/numKernels);i++)
		if(i<count)
			d_indexOrder[i]=i;
}


__global__ void sortAndBFS(int *d_x,int *d_y, int *d_z,int *d_indexOrder, int *d_partition_begin, int *d_partition_last,int d_numPartitions, int count, int *d_labels,int *d_queue,int *d_front,int *d_rear,int *d_numGroups, int *d_neighbours)
{
	int begin, last,i,j,sortPos,x,y,z,numberOfLabels, indexOrder,index, front, rear,partitionIndex;
	
	int hashvalue,dx[6],dy[6],dz[6],tempHashValue,k;

	partitionIndex=blockIdx.x*blockDim.x+threadIdx.x;
	if(partitionIndex>=d_numPartitions)
		return;
	__shared__ int hashGlobalMemory[hashTableWidth];

	begin=d_partition_begin[partitionIndex];
	last=d_partition_last[partitionIndex];
	for(i=begin;i<=last;i++)
	{
		hashvalue=(d_x[i]+d_y[i]+d_z[i])%numHashPerThread;
		sortPos=binarySearch(d_x,d_y, d_z,begin,(i-1),hashvalue);
		x=d_x[i];
		y=d_y[i];
		z=d_z[i];
		indexOrder=d_indexOrder[i];
		for(j=i-1;j-numThreadsPerBlock>=sortPos;j=j-numThreadsPerBlock)
		{
			shiftPos(d_x,d_y,d_z,d_indexOrder,j);
		}
		for(;j>=sortPos;j--)
		{
			shiftPos(d_x,d_y,d_z,d_indexOrder,j);
		}
		d_x[sortPos]=x;
		d_y[sortPos]=y;
		d_z[sortPos]=z;
		d_indexOrder[sortPos]=indexOrder;

	}
	hashvalue=-1;
	for(i=partitionIndex*numHashPerThread;i<(partitionIndex+1)*numHashPerThread;i++)
	{
		hashGlobalMemory[i]=-1;
	}

	for(i=begin;i<=last;i++)
	{
		d_labels[i]=-1;
		tempHashValue=(d_x[i]+d_y[i]+d_z[i])%numHashPerThread;
		if(tempHashValue!=hashvalue)
		{
			hashGlobalMemory[partitionIndex*numHashPerThread+tempHashValue]=i;
			hashvalue=i;
		}

	}




	if(partitionIndex==0)
	{
		d_front[0]=-1;
		d_rear[0]=-1;
		d_numGroups[0]=0;
	}



	__syncthreads();

	for(i=0;i<count;i++)
	{
		if(partitionIndex==0)
		{
			if(d_labels[d_indexOrder[i]]==-1)
			{
				d_front[0]=0;
				d_rear[0]=0;
				d_numGroups[0]++;
				d_queue[d_rear[0]]=i;
			}
	
		}

		while(d_front[0]!=-1 || rear>front)
		{
			front=d_front[0];
			rear=d_rear[0];
			for(j=0;j<6;j++)
				d_neighbours[j]=-1;
			dx[0]=d_x[d_queue[front]]-1;
			dy[0]=d_y[d_queue[front]];
			dz[0]=d_z[d_queue[front]];
			dx[1]=d_x[d_queue[front]];
			dy[1]=d_y[d_queue[front]]-1;
			dz[1]=d_z[d_queue[front]];
			dx[2]=d_x[d_queue[front]]+1;
			dy[2]=d_y[d_queue[front]];
			dz[2]=d_z[d_queue[front]];
			dx[3]=d_x[d_queue[front]];
			dy[3]=d_y[d_queue[front]]+1;
			dz[3]=d_z[d_queue[front]];
			dx[4]=d_x[d_queue[front]];
			dy[4]=d_y[d_queue[front]];
			dz[4]=d_z[d_queue[front]]+1;
			dx[5]=d_x[d_queue[front]];
			dy[5]=d_y[d_queue[front]];
			dz[5]=d_z[d_queue[front]]-1;

			for(j=0;j<6;j++)
			{
				if((compare(dx[j],dy[j],dz[j],d_x[d_partition_begin[partitionIndex]],d_y[d_partition_begin[partitionIndex]],d_z[d_partition_begin[partitionIndex]])>=0)&&(compare(dx[j],dy[j],dz[j],d_x[d_partition_begin[partitionIndex]],d_y[d_partition_begin[partitionIndex]],d_z[d_partition_begin[partitionIndex]])<=0))
				{
					if(hashGlobalMemory[partitionIndex*numHashPerThread+((dx[j]+dy[j]+dz[j])%numHashPerThread)]!=-1)
					{
						for(k=hashGlobalMemory[partitionIndex*numHashPerThread+((dx[j]+dy[j]+dz[j])%numHashPerThread)];((d_x[k]+d_y[k]+d_z[k])%numHashPerThread)==((dx[j]+dy[j]+dz[j])%numHashPerThread);k++)
						{
							if(compare(d_x[k],d_y[k],d_z[k],dx[j],dy[j],dz[j])==0)
							{
								if(d_labels[d_indexOrder[k]]==-1)
								{
									d_neighbours[j]=k;
									d_labels[d_indexOrder[k]]=d_numGroups[0];
								}
								break;
							}
							if(k==count-1)
								break;

						}
					}
				}
			}
			__syncthreads();
			if(partitionIndex==0)
			{
				for(k=0;k<6;k++)
				{
					if(d_neighbours[k]!=-1)
					{
						d_rear[0]++;
						d_queue[d_rear[0]]=d_neighbours[k];
					}
				}
				d_front[0]++;
			}
			__syncthreads();
		}
	}

}


int main()
{
	int count,i,*d_front,*d_rear,h_front,h_rear,*h_labels,*d_labels,*h_partition_begin,*d_partition_begin, *orderedGroups;
	int *h_partition_last,*d_partition_last,*d_queue, *h_numGroups, *d_numGroups,*d_indexOrder,*d_neighbours;
	int nx,ny,nz,x,y,z,*h_x,*h_y,*h_z,*d_x,*d_y,*d_z,*h_numPartitions,*d_numPartitions,groupLabelJockey;
	FILE *fp, *ofp;

	fp=fopen("data.txt","r");
	count=0;
	fscanf(fp,"Nx=%d Ny=%d Nz=%d",&nx,&ny,&nz);
	printf("%d %d %d\n",nx,ny,nz);
	while(feof(fp)==0)
	{
		fscanf(fp,"%d %d %d\n",&x,&y,&z);
		printf("%d %d %d\n",x,y,z);
		count++;
	}
	printf("%d\n",count);
	fclose(fp);
	h_x=(int*)malloc(sizeof(int)*count);
	h_y=(int*)malloc(sizeof(int)*count);
	h_z=(int*)malloc(sizeof(int)*count);
	h_partition_begin=(int*)malloc(sizeof(int));
	h_partition_last=(int*)malloc(sizeof(int));
	h_labels=(int*)malloc(sizeof(int)*count);
	h_numPartitions=(int*)malloc(sizeof(int));
	h_numGroups=(int*)malloc(sizeof(int));

	hipMalloc(&d_x,count*sizeof(int));
	hipMalloc(&d_y,count*sizeof(int));
	hipMalloc(&d_z,count*sizeof(int));
	hipMalloc(&d_partition_begin,numKernels*sizeof(int));
	hipMalloc(&d_partition_last,numKernels*sizeof(int));
	hipMalloc(&d_queue,sizeof(int)*count);
	hipMalloc(&d_labels,sizeof(int)*count);
	hipMalloc(&d_front,sizeof(int));
	hipMalloc(&d_rear,sizeof(int));
	hipMalloc(&d_labels,sizeof(int)*count);
	hipMalloc(&d_numPartitions,sizeof(int));
	hipMalloc(&d_numGroups,sizeof(int));
	hipMalloc(&d_indexOrder, sizeof(int)*count);
	hipMalloc(&d_neighbours,sizeof(int)*6); /*0:left, 1: behind, 2:right,3: front, 4: top, 5:bottom*/


	fp=fopen("data.txt","r");
	fscanf(fp,"Nx=%d Ny=%d Nz=%d",&nx,&ny,&nz);
	printf("%d %d %d\n",nx,ny,nz);
	for(i=0;i<count;i++)
	fscanf(fp,"%d %d %d\n",&h_x[i],&h_y[i],&h_z[i]);
	
	fclose(fp);

	h_partition_begin[0]=0;
	h_partition_last[0]=count-1;

	hipMemcpy(d_x,h_x,count*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_y,h_y,count*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_z,h_z,count*sizeof(int),hipMemcpyHostToDevice);
	//cudaMemcpy(d_front,h_front,sizeof(int),cudaMemcpyHostToDevice);
	//cudaMemcpy(d_rear,h_rear,sizeof(int),cudaMemcpyHostToDevice);
	//cudaMemcpy(d_rear,h_rear,sizeof(int),cudaMemcpyHostToDevice);


	setIndexOrder<<<numKernels,1>>>(d_indexOrder,count);

	initializePartitionTable<<<numKernels,1>>>(d_partition_begin,d_partition_last);
	
	hipMemcpy(d_partition_begin,h_partition_begin,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_partition_last,h_partition_last,sizeof(int),hipMemcpyHostToDevice);
	for(i=1;i<=(numKernels/2);i=i*2)
	{

		createPartition<<<1,i>>>(count,d_x,d_y,d_z,d_indexOrder,d_partition_begin,d_partition_last);

	}

	countPartitions<<<1,1>>>(d_partition_begin,d_partition_last,d_numPartitions);

	hipMemcpy(h_numPartitions,d_numPartitions,sizeof(int),hipMemcpyDeviceToHost);

	sortAndBFS<<<1,numKernels>>>(d_x,d_y,d_z,d_indexOrder,d_partition_begin,d_partition_last,h_numPartitions[0],count,d_labels,d_queue,d_front,d_rear,d_numGroups,d_neighbours);

	hipMemcpy(h_labels,d_labels,count*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(h_numGroups,d_numGroups,sizeof(int),hipMemcpyDeviceToHost);
	orderedGroups=(int*)malloc(sizeof(int)*(h_numGroups[0]+1));
	for(i=1;i<=h_numGroups[0];i++)
		orderedGroups[i]=-1;
	groupLabelJockey=0;
	for(i=0;i<count;i++)
	if(orderedGroups[h_labels[i]]==-1)
	{
		groupLabelJockey++;
		orderedGroups[h_labels[i]]=groupLabelJockey;
	}
	ofp=fopen("result.txt","w");
	fprintf(ofp,"Nx=%d Ny=%d  Nz=%d Cluster=%d\nX Y Z id\n",nx,ny,nz,h_numGroups[0]);
	for(i=0;i<count;i++)
		fprintf(ofp,"%d %d %d %d\n",h_x[i],h_y[i],h_z[i],orderedGroups[h_labels[i]]);
	fclose(ofp);

	free(h_labels);
	free(h_x);
	free(h_y);	
	free(h_z);
	free(h_partition_begin);
	free(h_partition_last);
	
	hipFree(d_front);
	hipFree(d_rear);
	hipFree(d_labels);
	hipFree(d_partition_begin);
	hipFree(d_partition_last);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	hipFree(d_queue);
	hipFree(d_numGroups);
	hipFree(d_indexOrder);
	hipFree(d_neighbours);
	
	return 0;
}
